#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <ctime>
#include <cstdlib>
#include <math.h>

using namespace std;

//======================network architecture=====================

const int numInputs = 3;         // Input nodes, plus the bias input.
const int numHidden = 100;
const int numHidden2 = 100;
const int numOutput = 1;

//========================device copies of network architecture===========
__device__ __constant__ int numInputs = 3;
__device__ __constant__ int numHidden = 100;
__device__ __constant__ int numHidden2 = 100;
__device__ __constant__ int numOutput = 1;
//=========================device copies network parameters=====================                             
__device__ __constant__ double LR = 0.5;       // Learning rate                   
__device__ __constant__ double M = 0.1;      // Momentum Rate

//=========================dataset variables======================
const int numPatterns = 100; // number of input patterns for circle experiment.
const int radius = 1;          //radius of the circle
const int minRange = -2;       //input range for points in the dataset
const int maxRange = 2;
const int numEpochs = 1000;    //Amount of training to do. epoch = 1 exposure to the entire training set

int patNum = 0;                 //tracking the pattern number
double Guess = 0.0;             // network output value.
double errThisPat = 0.0;
double RMSerror = 0.0;      // Squared error
double errorTotal = 0.0;
double SumOf = 0;

//==================Matrices Vital for Backprop================
//===================================================NEED TO BE COPIED TO DEVICE IN BACKPROPPREP FUNCTION========================================================
double hiddenActivation[numHidden] = { 0.0 };
double hidden2Activation[numHidden2] = { 0.0 };
double outputActivation[numOutput] = { 0.0 };

double weightsIH[numInputs][numHidden]; // Input to Hidden weights.
double weightsH1H2[numHidden][numHidden2];
double weightsHO[numHidden2][numOutput]; // Hidden to Output weights.

double outputDeltas[numOutput];
double hidden2Deltas[numHidden2];
double hiddenDeltas[numHidden];


double changeHidden[numInputs][numHidden] = { 0.0 };
double changeHidden2[numHidden][numHidden2] = { 0.0 };
double changeOutput[numHidden2][numOutput] = { 0.0 };
//----------------------------------------------------

double trainInputs[numPatterns][numInputs];
int trainOutput[numPatterns];           // "Actual" output values.

//=============================Function Prototypes===================
void train();
void initWeights();
void feedforward();
void backProp();
void initData();
void test_1();
double getRand(double num1, double num2);
double sigmoid(double x);
double dsigmoid(double x);
hipError_t CudaBackProp();

__global__ void CudaBackProp_part1()
{
    int i = threadIdx.x;
	outputDeltas[i] = LR * errThisPat * outputActivation[i];
}

__global__ void CudaBackProp_part2()
{
	int j = threadIdx.x;

	errors = 0;
	for (int k = 0; k < numOutput; k++)
	{
		errors = errors + outputDeltas[k] * weightsHO[j][k];
	}
	hidden2Deltas[j] = dsigmoid(hidden2Activation[j]) * errors;
}

__global__ void CudaBackProp_part3()
{
	int j = threadIdx.x;
	errors = 0;
	for (int k = 0; k < numHidden2; k++)
	{
		errors = errors + hidden2Deltas[k] * weightsH1H2[j][k];
	}
	hiddenDeltas[j] = dsigmoid(hiddenActivation[j]) * errors;
}

__global__ void CudaBackProp_part4()
{
	int j = threadIdx.x;
	for (int k = 0; k< numOutput; k++)
	{
		change = outputDeltas[k] * hidden2Activation[j];
		weightsHO[j][k] = weightsHO[j][k] + LR*change + M*changeOutput[j][k];
		changeOutput[j][k] = change;
	}
}

__global__ void CudaBackProp_part5()
{
	int j = threadIdx.x;
	for (int k = 0; k< numHidden2; k++)
	{
		change = hidden2Deltas[k] * hiddenActivation[j];
		weightsH1H2[j][k] = weightsH1H2[j][k] + LR*change + M*changeHidden2[j][k];
		changeHidden2[j][k] = change;
	}
}

__global__ void CudaBackProp_part6()
{
	int j = threadIdx.x;
	for (int k = 0; k< numHidden; k++)
	{
		change = hiddenDeltas[k] * trainInputs[patNum][j];
		changeHidden[j][k] = change;
		weightsIH[j][k] = weightsIH[j][k] + LR*change + M*changeHidden[j][k];
	}
}

int main()
{   
	srand((unsigned)time(0));   // Seed the random num generator
	initWeights();
	cout << "created weights" << endl;

	initData();
	cout << "created Training data and starting training" << endl;

	train();
	cout << "Finished training" << endl;

	//Training has finished.
	cout << "testing" << endl;
	test_1();
	
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	//----------------------------------------------------------------------
    return 0;
}

// Helper function for using CUDABackProp
//=============================NEEDS WORK==================================
hipError_t CudaBackPropPrep(int numInputs,
							 double hiddenActivation[], int numHidden, 
							 double Hidden2Activation[], int numHidden2,
							 double outputActivation[], int numOutput,
							 double weightsIH[][], double weightsH1H2[][], double weightsHO[][], 
							 double outputDeltas[], double hiddenDeltas[], double hidden2Deltas[],
							 double changeHidden[][], double changeHidden2[][], double changeOutput[][]
							)
{
	hipError_t cudaStatus;
	//---------------------device versions of vectors---------------------
    double *dev_HiddenActivation = 0;
    double *dev_Hidden2Activation = 0;
	double *dev_outputActivation = 0;

	double *dev_weightsIH = 0;
	double *dev_weightsH1H2 = 0;
	double *dev_weightsHO = 0;

	double *dev_outputDeltas = 0;
	double *dev_hiddenDeltas = 0;
	double *dev_hidden2Deltas = 0;

	double *dev_changeHidden = 0;
	double *dev_changeHidden2 = 0;
	double *dev_changeOutput = 0;
	//---------------------------------------------------------------------------

    // Choose which GPU to run on
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // -----------------------------------Allocate 1D GPU buffers for vectors-----------------------------------
    cudaStatus = hipMalloc((void**)&dev_HiddenActivation, numHidden * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_Hidden2Activation, numHidden2 * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_OutputActivation, numOutput * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_outputDeltas, numOutput * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_HiddenDeltas, numHidden * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	
	cudaStatus = hipMalloc((void**)&dev_Hidden2Deltas, numHidden2 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//------------------------------------Allocate 2D GPU buffers for 2D vectors--------------------------------------
	//T* pElement = (T*)((char*)BaseAddress + Row * pitch) + Column; <---- addressing a "ptich"
	size_t pitch_WeightsIH;
	cudaStatus = hipMallocPitch((void **)&dev_weightsIH,
		&pitch_WightsIH,
		numInputs * sizeOf(double),
		numHidden * sizeOf(double)
	)
	if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

	size_t pitch_WeightsH1H2;
	cudaStatus = hipMallocPitch((void **)&dev_weightsH1H2,
		&pitch_WeightsH1H2,
		numHidden * sizeOf(double),
		numHidden2 * sizeOf(double)
		)
	if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

	size_t pitch_WeightsHO;
	cudaStatus = hipMallocPitch((void **)&dev_weightsHO,
		pitch_WeightsHO,
		numHidden2 * sizeOf(double),
		numOutput * sizeOf(double)
		)
	if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

	cudaStatus = hipMallocPitch((void **)&dev_weightsIH,
			size_t * pitch,
			size_t 	width,
			size_t 	height
		)
	if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

	size_t pitch_changeHidden;
	cudaStatus = hipMallocPitch((void **)&dev_weightsIH,
			size_t * pitch,
			size_t 	width,
			size_t 	height
		)
	if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

	size_t pitch_changeHidden2;
	cudaStatus = hipMallocPitch((void **)&dev_weightsIH,
			size_t * pitch,
			size_t 	width,
			size_t 	height
		)
	if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

	size_t pitch_changeOutput;
	cudaStatus = hipMallocPitch((void **)&dev_weightsIH,
		size_t * pitch,
		size_t 	width,
		size_t 	height
	)
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

	//====================================================================================================================================

    //------------------------------------------Copy input vectors from host memory to GPU buffers---------------------------------------
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//====================================================================================================================================
	return cudaStatus;
    
}
//--------------------------------------------------------
void train()
{
	hipError_t cudaStatus = cudaBackPropprep();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cuda failed!");
		return 1;
	}
	for (int j = 0; j <= numEpochs; j++)
	{
		errorTotal = 0;
		for (int i = 0; i < numPatterns; i++)
		{
			patNum = i;
			//Calculate the output and error for this pattern.
			feedforward();
			errorTotal = errorTotal + RMSerror;
			
			//-----------------------Launch kernels on the GPU with one thread for each element in outside array----------------
			//replace size with length of outer array in nested loop.
			CudaBackProp_part1 <<<1, numOutput >>>();
			__syncthreads();
			CudaBackProp_part2 << <1, numHidden2 >>>();
			__syncthreads();
			CudaBackProp_part3 <<<1, numHidden >>>();
			__syncthreads();
			CudaBackProp_part4 <<<1, numHidden2 >>>();
			__syncthreads();
			CudaBackProp_part5 <<<1, numHidden >>>();
			__syncthreads();
			CudaBackProp_part6 << <1, numInputs >> >();
			__syncthreads();

			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
				goto Error;
			}


		}
		errorTotal = sqrt(errorTotal / numPatterns);
		if (j % 100 == 0) { cout << "epoch = " << j << " Error = " << errorTotal << endl; }
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	//-------------------------Need to do this for all weights vectors-------------------------
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//-----------------------------------------------------------------------------------------
	Error:
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);

	return cudaStatus;
	
}

void initWeights()
{
	// Initialize weights to random values.
	for (int j = 0; j < numHidden; j++)
	{
		for (int i = 0; i < numInputs - 1; i++)
		{
			weightsIH[i][j] = getRand(-.2, .2);

		}
		weightsIH[j][numInputs - 1] = 1;//Bias
	}
	for (int j = 0; j < numHidden2; j++)
	{
		for (int i = 0; i < numHidden; i++)
		{
			weightsH1H2[i][j] = getRand(-0.4, 0.4);
		}
	}
	for (int j = 0; j < numOutput; j++)
	{
		for (int i = 0; i < numHidden2; i++)
		{
			weightsHO[i][j] = getRand(-0.4, 0.4);
		}
	}
}

void initData()
{
	//Training set of points inside/outside a circle
	for (int j = 0; j < numPatterns; j++)
	{
		for (int i = 0; i < numInputs; i++)
		{
			trainInputs[i][j] = getRand(minRange, maxRange);
		}

		if (((trainInputs[j][0] * trainInputs[j][0]) + (trainInputs[j][1] * trainInputs[j][1])) < (radius*radius))
		{//if in circle answer is 1
			trainOutput[j] = 1;
		}//otherwise answer is 0
		else { trainOutput[j] = 0; }
	}
}

void feedforward()
{
	//input nodes dont have activations, except an assignment from raw to an array

	//hidden nodes activations
	for (int i = 0; i < numHidden; i++)
	{
		SumOf = 0.0;
		for (int j = 0; j < numInputs; j++)
		{
			SumOf = SumOf + (trainInputs[patNum][j] * weightsIH[j][i]);
		}
		hiddenActivation[i] = sigmoid(SumOf);
	}

	for (int i = 0; i < numHidden2; i++)
	{
		SumOf = 0.0;
		for (int j = 0; j < numHidden; j++)
		{
			SumOf = SumOf + (hiddenActivation[j] * weightsH1H2[j][i]);
		}
		hidden2Activation[i] = sigmoid(SumOf);
	}

	//output nodes activations
	for (int i = 0; i < numOutput; i++)
	{
		SumOf = 0.0;
		for (int j = 0; j < numHidden2; j++)
		{
			SumOf = SumOf + (hidden2Activation[j] * weightsHO[j][i]);
		}
		outputActivation[i] = sigmoid(SumOf);
	}

	//Author Note:Needs general case for more than 1 output
	Guess = outputActivation[0];
	errThisPat = trainOutput[patNum] - Guess;
	RMSerror = errThisPat * errThisPat;

}

void test_1()
{
	for (int i = 0; i < numPatterns / 10; i++)
	{
		patNum = i;
		feedforward();
		cout << "pattern = " << patNum + 1 <<
			"| actual answer = " << trainOutput[patNum] <<
			"| Neural Net guess = " << Guess << endl;
	}
}

double getRand(double num1, double num2)
{
	return double((num1 - num2)*(rand() / double(RAND_MAX)) + num1); //(b-a)*random.random() + a
}

double sigmoid(double x)
{
	return 1 / (1 + exp(-x));
}

__global__ double sigmoid(double x)
{
	return 1 / (1 + exp(-x));
}

__device__ double dsigmoid(double x)
{
	return sigmoid(x) * (1 - sigmoid(x));
}

void test_2()
{
	int counter = 0;
	for (int i = 0; i < numPatterns / 10; i++)
	{
		patNum = i;
		feedforward();


		if (Guess >= .5 && trainOutput[patNum] == 1)
		{
			counter += 1;
		}
		else if (Guess < .5 && trainOutput[patNum] == -1)
		{
			counter += 1;
		}
	}
	cout << ((counter / (numPatterns / 10)) * 100) << "% classification accuracy" << endl;
}
