
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <cmath>
#include <ctime>
#include <cstdlib>
#include <math.h>

using namespace std;

//======================network architecture=====================

const int numInputs = 3;         // Input nodes, plus the bias input.
const int numHidden = 100;       //number of nodes in hidden layer 1
const int numHidden2 = 100;		//number of nodes in hidden layer 2
const int numOutput = 1;		//number of output nodes

//========================device copies of network architecture===========
__device__ __constant__ int dev_numInputs = 3;
__device__ __constant__ int dev_numHidden = 100;
__device__ __constant__ int dev_numHidden2 = 100;
__device__ __constant__ int dev_numOutput = 1;
//=========================device copies network parameters=====================                             
__device__ __constant__ double LR = 0.5;       // Learning rate                   
__device__ __constant__ double M = 0.1;      // Momentum Rate

//=========================dataset variables======================
const int numPatterns = 100; // number of input patterns for circle experiment.
const int radius = 1;          //radius of the circle
const int minRange = -2;       //input range for points in the dataset
const int maxRange = 2;
const int numEpochs = 1000;    //Amount of training to do. epoch = 1 exposure to the entire training set


__device__ double dev_Guess = 0.0;             // network output value.
__device__ double dev_errThisPat = 0.0;
__device__ double dev_RMSerror = 0.0;      // Squared error
__device__ double dev_errorTotal = 0.0;

 int patNum = 0;
 int *dev_patNum; //tracking the pattern number
 double Guess = 0.0;             // network output value.
 double errThisPat = 0.0;
 double RMSerror = 0.0;      // Squared error
 double errorTotal = 0.0;
 double SumOf = 0;

 double **trainInputs;
 double *trainOutput = new double[numPatterns];           // "Actual" output values.




//==================Matrices Vital for Backprop================
//may need to malloc
double *hiddenActivation = new double[numHidden];
double *hidden2Activation= new double[numHidden2];
double *outputActivation = new double[numOutput];

double **weightsIH = (double**)malloc((numHidden * numInputs) * sizeof(double)); // Input to Hidden weights.
double **weightsH1H2 = (double**)malloc((numHidden * numHidden2) * sizeof(double));
double **weightsHO = (double**)malloc((numOutput * numHidden2) * sizeof(double)); // Hidden to Output weights.

double *outputDeltas=new double[numOutput];
double *hidden2Deltas=new double[numHidden2];
double *hiddenDeltas=new double[numHidden];


double **changeHidden = (double**)malloc((numHidden * numInputs) *sizeof(double));
double **changeHidden2 = (double**)malloc((numHidden * numHidden2) *sizeof(double));
double **changeOutput = (double**)malloc((numOutput * numHidden2) *sizeof(double));

//=================device versions of vectors=================
double *dev_hiddenActivation = 0;
double *dev_hidden2Activation = 0;
double *dev_outputActivation = 0;

double **dev_weightsIH = 0;
double **dev_weightsH1H2 = 0;
double **dev_weightsHO = 0;

double *dev_outputDeltas = 0;
double *dev_hiddenDeltas = 0;
double *dev_hidden2Deltas = 0;

double **dev_changeHidden = 0;
double **dev_changeHidden2 = 0;
double **dev_changeOutput = 0;

double **dev_trainInputs = 0;
double *dev_trainOutput = 0;
//========================pitch sizes for 2D vector memory allocation=============================
size_t pitch_trainInputs;
size_t pitch_WeightsIH;
size_t pitch_WeightsH1H2;
size_t pitch_WeightsHO;
size_t pitch_changeHidden;
size_t pitch_changeHidden2;
size_t pitch_changeOutput;


//===================================Function Prototypes===================

void initWeights();
void feedforward();
void backProp();
hipError_t initData();
//void test_1();
double getRand(double num1, double num2);
double sigmoid(double x);
double dsigmoid(double x);
hipError_t DeviceMemoryPrep(int numInputs,
	double hiddenActivation[], int numHidden,
	double hidden2Activation[], int numHidden2,
	double outputActivation[], int numOutput,
	double **weightsIH, double **weightsH1H2, double **weightsHO,
	double outputDeltas[], double hiddenDeltas[], double hidden2Deltas[],
	double **changeHidden, double **changeHidden2, double **changeOutput);
hipError_t train();
__global__ void CudaBackProp_part1(double dev_outputDeltas[], double dev_outputActivation[]);
__global__ void CudaBackProp_part2(double dev_outputDeltas[], double**dev_weightsHO, double dev_hidden2Deltas[], double dev_hidden2Activation[]);
__global__ void CudaBackProp_part3(double dev_Hidden2Deltas[], double**dev_weightsH1H2, double dev_hiddenDeltas[], double hiddenActivation[]);
__global__ void CudaBackProp_part4(double dev_outputDeltas[], double dev_hidden2Activation[], double**dev_weightsHO, double**dev_changeOutput);
__global__ void CudaBackProp_part5(double dev_hidden2Deltas[], double dev_hiddenActivation[], double**dev_weightsH1H2, double**dev_changeHidden2);
__global__ void CudaBackProp_part6(int *patnum, double dev_hiddenDeltas[], double** dev_trainInputs, double**dev_changeHidden, double**dev_weightsIH);
__global__ void CudaFeedForward_part1(int *patnum, double**dev_trainInputs, double**dev_weightsIH, double dev_hiddenActivation[]);
__global__ void CudaFeedForward_part2( double dev_hiddenActivation[], double**dev_weightsH1H2, double dev_hidden2Activation[]);
__global__ void CudaFeedForward_part3(int *patnum, double dev_hidden2Activation[], double**dev_weightsHO, double dev_outputActivation[], double dev_trainOutput[]);


int main()
{   
	hipError_t cudaStatus;
	//srand((unsigned)time(0));   // Seed the random num generator
	initWeights();
	cout << "created weights" << endl;

	initData();
	cout << "created Training data" << endl;

	cudaStatus = train();
	if (cudaStatus != hipSuccess) {
		cout << stderr << " Training Function Failed" << endl;
		return 1;
	}
	cout << "Finished training" << endl;

	//Training has finished.
	cout << "testing" << endl;
	//test_1();
	
    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	//----------------------------------------------------------------------
	free(weightsIH);
	free(weightsH1H2);
	free(weightsHO);
	free(changeHidden);
	free(changeOutput);
	free(changeHidden2);



    return 0;
}

/* Helper function for Memory Allocation.
INPUT:Takes all backpropagation arrays as inputs, aswell as variables describing dimensions.
OUTPUT: outputs a cudaError_t type, describing any errors that may have happened during operation.
FUNCTIONALITY: allocates memory on the device for all relevant arrays for backprop and feedforward functions.
*/
hipError_t DeviceMemoryPrep(int numInputs,
							 double hiddenActivation[], int numHidden, 
							 double hidden2Activation[], int numHidden2,
							 double outputActivation[], int numOutput,
							 double **weightsIH, double **weightsH1H2, double **weightsHO, 
							 double outputDeltas[], double hiddenDeltas[], double hidden2Deltas[],
							 double **changeHidden, double **changeHidden2, double **changeOutput
							)
{
	hipError_t cudaStatus;

    // =====================================Choose which GPU to run on============================================
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    //=========================================Allocate 1D GPU buffers for vectors==========================================
    cudaStatus = hipMalloc((void**)&dev_hiddenActivation, numHidden * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMalloc1 failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_hidden2Activation, numHidden2 * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMalloc2 failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_outputActivation, numOutput * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMalloc3 failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_outputDeltas, numOutput * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMalloc4 failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_hiddenDeltas, numHidden * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMalloc5 failed!");
		goto Error;
	}
	
	cudaStatus = hipMalloc((void**)&dev_hidden2Deltas, numHidden2 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMalloc6 failed!");
		goto Error;
	}

	//========================================Allocate 2D GPU buffers for 2D vectors==========================================
	//T* pElement = (T*)((char*)BaseAddress + Row * pitch) + Column; <---- addressing a "ptich"
	
	cudaStatus = hipMallocPitch((void **)&dev_weightsIH,
		&pitch_WeightsIH,
		numInputs * sizeof(double),
		numHidden * sizeof(double)
	);
	if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cudaMallocPitch1 failed!");
			goto Error;
		}

	
	cudaStatus = hipMallocPitch((void **)&dev_weightsH1H2,
		&pitch_WeightsH1H2,
		numHidden * sizeof(double),
		numHidden2 * sizeof(double)
	);
	if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cudaMallocPitch2 failed!");
			goto Error;
		}

	
	cudaStatus = hipMallocPitch((void **)&dev_weightsHO,
		&pitch_WeightsHO,
		numHidden2 * sizeof(double),
		numOutput * sizeof(double)
	);
	if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cudaMallocPitch3 failed!");
			goto Error;
		}

	
	cudaStatus = hipMallocPitch((void **)&dev_changeHidden,
		&pitch_changeHidden,
		numInputs * sizeof(double),
		numHidden * sizeof(double)
	);
	if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cudaMallocPitch4 failed!");
			goto Error;
		}

	
	cudaStatus = hipMallocPitch((void **)&dev_changeHidden2,
		&pitch_changeHidden2,
		numHidden * sizeof(double),
		numHidden2 * sizeof(double)
	);
	if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cudaMallocPitch5 failed!");
			goto Error;
		}

	
	cudaStatus = hipMallocPitch((void **)&dev_changeOutput,
		&pitch_changeOutput,
		numHidden2 * sizeof(double),
		numOutput * sizeof(double)
	);
	if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cudaMallocPitch6 failed!");
			goto Error;
		}

	//-------------------------------------------------------------------------------------

    //=======================================Copy 1D vectors from host memory to GPU buffers======================================
    cudaStatus = hipMemcpy(dev_hiddenActivation, hiddenActivation, numHidden * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMemcpy1 failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_hidden2Activation, hidden2Activation, numHidden2 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMemcpy2 failed!");
        goto Error;
    }
	cudaStatus = hipMemcpy(dev_outputActivation, outputActivation, numOutput * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpy3 failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_outputDeltas, outputDeltas, numOutput * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpy4 failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_hiddenDeltas, hiddenDeltas, numHidden * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpy5 failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_hidden2Deltas, hidden2Deltas, numHidden2 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMemcpy6 failed!");
		goto Error;
	}

	//===========================================Copy 2D vectors=============================================
	cudaStatus = hipMemcpy2D(dev_weightsIH, pitch_WeightsIH*sizeof(double), weightsIH, pitch_WeightsIH,
		numInputs * sizeof(double), numHidden, hipMemcpyHostToDevice);
	if(cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D 1 failed!");
		goto Error;
	}
	
	cudaStatus = hipMemcpy2D(dev_weightsH1H2, pitch_WeightsH1H2 * sizeof(double), weightsH1H2, pitch_WeightsH1H2,
		numHidden * sizeof(double),numHidden2 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D 2 failed!");
		goto Error;
	}
	
	cudaStatus = hipMemcpy2D(dev_weightsHO, pitch_WeightsHO * sizeof(double), weightsHO, pitch_WeightsHO,
		numHidden2 * sizeof(double),numOutput * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D 3 failed!");
		goto Error;
	}
	
	cudaStatus = hipMemcpy2D(dev_changeHidden, pitch_changeHidden * sizeof(double), changeHidden, pitch_changeHidden,
		numInputs * sizeof(double),numHidden * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D 4 failed!");
		goto Error;
	}
	
	cudaStatus = hipMemcpy2D(dev_changeHidden2, pitch_changeHidden2 * sizeof(double), changeHidden2,pitch_changeHidden2,
		numHidden * sizeof(double),numHidden2 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D 5 failed!");
		goto Error;
	}
	
	cudaStatus = hipMemcpy2D(dev_changeOutput, pitch_changeOutput * sizeof(double), changeOutput, pitch_changeOutput,
		numHidden2 * sizeof(double),numOutput * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D 6 failed!");
		goto Error;
	}

Error:
	hipFree(dev_hiddenActivation);
	hipFree(dev_hidden2Activation);
	hipFree(dev_outputActivation);

	hipFree(dev_weightsIH);
	hipFree(dev_weightsH1H2);
	hipFree(dev_weightsHO);

	hipFree(dev_outputDeltas);
	hipFree(dev_hiddenDeltas);
	hipFree(dev_hidden2Deltas);

	hipFree(dev_changeHidden);
	hipFree(dev_changeHidden2);
	hipFree(dev_changeOutput);



	//----------------------------------------------------------------------------------------------------------------
	return cudaStatus;
    
}

hipError_t train()
{
	hipError_t cudaStatus = DeviceMemoryPrep(numInputs,
		hiddenActivation, numHidden,
		hidden2Activation, numHidden2,
		outputActivation, numOutput,
		weightsIH, weightsH1H2, weightsHO,
		outputDeltas, hiddenDeltas, hidden2Deltas,
		changeHidden, changeHidden2, changeOutput);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "DeviceMemoryPrep failed!");
		return cudaStatus;
	}
	
	for (int j = 0; j <= numEpochs; j++)
	{
		errorTotal = 0;
		for (int i = 0; i < numPatterns; i++)
		{
			patNum = i;
			hipMalloc((void**)&dev_patNum, sizeof(int));
			hipMemcpy(dev_patNum, &patNum, sizeof(int), hipMemcpyHostToDevice);
			//Calculate the output and error for this pattern.
			feedforward();
			backProp();
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
		}
		errorTotal = sqrt(errorTotal / numPatterns);
		if (j % 100 == 0) { cout << "epoch = " << j << " Error = " << errorTotal << endl; }
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	//-------------------------Copy output weight vectors from GPU buffer to host memory.-------------------------

	cudaStatus = hipMemcpy2D(weightsIH, numHidden * sizeof(double), dev_weightsIH, pitch_WeightsIH,
		numInputs * sizeof(double), numHidden, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D return failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy2D(weightsH1H2, numHidden * sizeof(double), dev_weightsH1H2, pitch_WeightsH1H2,
		numHidden * sizeof(double), numHidden2 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D return failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy2D(weightsHO, numHidden2 * sizeof(double), dev_weightsHO, pitch_WeightsHO,
		numHidden2 * sizeof(double), numOutput * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D return failed!");
		goto Error;
	}

	//-----------------------------------------------------------------------------------------
Error:
	hipFree(dev_hiddenActivation);
	hipFree(dev_hidden2Activation);
	hipFree(dev_outputActivation);

	hipFree(dev_weightsIH);
	hipFree(dev_weightsH1H2);
	hipFree(dev_weightsHO);

	hipFree(dev_outputDeltas);
	hipFree(dev_hiddenDeltas);
	hipFree(dev_hidden2Deltas);

	hipFree(dev_changeHidden);
	hipFree(dev_changeHidden2);
	hipFree(dev_changeOutput);
	return cudaStatus;
	
}

void backProp()
{
	
	CudaBackProp_part1<<<1, numOutput >>>(dev_outputDeltas, dev_outputActivation);
	
	CudaBackProp_part2<<<1, numHidden2 >>>(dev_outputDeltas, dev_weightsHO, dev_hidden2Deltas, dev_hidden2Activation);
	
	CudaBackProp_part3<<<1, numHidden >>>(dev_hidden2Deltas, dev_weightsH1H2, dev_hiddenDeltas,  hiddenActivation);
	
	CudaBackProp_part4<<<1, numHidden2 >>>(dev_outputDeltas, dev_hidden2Activation, dev_weightsHO, dev_changeOutput);
	
	CudaBackProp_part5<<<1, numHidden >>>(dev_hidden2Deltas, dev_hiddenActivation, dev_weightsH1H2, dev_changeHidden2);
	
	CudaBackProp_part6<<<1, numInputs >>>(dev_patNum, dev_hiddenDeltas,  dev_trainInputs ,dev_changeHidden, dev_weightsIH);
	
}

void initWeights()
{
	//=====================================initialise 2d arrays=========================================
	weightsIH = new double*[numInputs];
	for (int i = 0; i < numInputs; i++)
	{
		weightsIH[i] = new double[numHidden];
	}

	weightsH1H2 = new double*[numHidden];
	for (int i = 0; i < numHidden; i++)
	{
		weightsH1H2[i] = new double[numHidden2];
	}

	weightsHO = new double*[numHidden2];
	for (int i = 0; i < numHidden2; i++)
	{
		weightsH1H2[i] = new double[numOutput];
	}

	changeHidden = new double*[numInputs];
	for (int i = 0; i < numInputs; i++)
	{
		changeHidden[i] = new double[numHidden];
	}

	changeHidden2 = new double*[numHidden];
	for (int i = 0; i < numHidden; i++)
	{
		changeHidden2[i] = new double[numHidden2];
	}

	changeOutput = new double*[numHidden2];
	for (int i = 0; i < numHidden2; i++)
	{
		changeOutput[i] = new double[numOutput];
	}
	//=================================Initialize weights to random values.======================================
	for (int j = 0; j < numHidden; j++)
	{
		for (int i = 0; i < numInputs - 1; i++)
		{
			weightsIH[i][j] = getRand(-.2, .2);
		}
		weightsIH[numInputs - 1][j] = 1;//Bias
	}
	for (int j = 0; j < numHidden2; j++)
	{
		for (int i = 0; i < numHidden; i++)
		{
			weightsH1H2[i][j] = getRand(-0.4, 0.4);
		}
	}
	for (int j = 0; j < numOutput; j++)
	{
		for (int i = 0; i < numHidden2; i++)
		{
			weightsHO[i][j] = getRand(-0.4, 0.4);
		}
	}
}

hipError_t initData()
{
	hipError_t cudaStatus;
	trainInputs = new double*[numPatterns];
	for (int i = 0; i < numPatterns; i++)
	{
		trainInputs[i] = new double[numInputs];
	}
	//data set of points inside/outside a circle
	for (int j = 0; j < numPatterns; j++){
		for (int i = 0; i < numInputs; i++){
			trainInputs[i][j] = getRand(minRange, maxRange);
		}

		if (((trainInputs[j][0] * trainInputs[j][0]) + (trainInputs[j][1] * trainInputs[j][1])) < (radius*radius)){//if in circle answer is 1
			trainOutput[j] = 1;
		}//otherwise answer is 0
		else { 
			trainOutput[j] = 0; 
		}
	}

	//==============================allocate and copy memory for output vector===========================
	cudaStatus = hipMalloc((void**)&dev_trainOutput, numPatterns * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc output vec failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_trainOutput, dev_trainOutput, numPatterns * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy output vec failed!");
		goto Error;
	}

	//===========================allocate and copy memory for input matrix============================
	cudaStatus = hipMallocPitch((void **)&dev_trainInputs,
		&pitch_trainInputs,
		numPatterns * sizeof(double),
		numInputs * sizeof(double)
	);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMallocPitch1 failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy2D(dev_trainInputs, pitch_trainInputs * sizeof(double), trainInputs, pitch_trainInputs,
		numPatterns * sizeof(double), numInputs, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy2D 1 failed!");
		goto Error;
	}
	
	
Error:
	hipFree(dev_trainInputs);
	hipFree(dev_trainOutput);

	return cudaStatus;

}

void feedforward()
{
	//input nodes dont have activations, except an assignment from raw to an array
	//hidden nodes activations
	CudaFeedForward_part1<<<1, numHidden >>>(dev_patNum, dev_trainInputs, dev_weightsIH,  dev_hiddenActivation);
	
	CudaFeedForward_part2<<<1, numHidden2 >>>(dev_hiddenActivation, dev_weightsH1H2,  dev_hidden2Activation);
	
	CudaFeedForward_part3<<<1, numOutput >>>(dev_patNum,dev_hidden2Activation, dev_weightsHO, dev_outputActivation, dev_trainOutput);
	
	//Author Note:Needs general case for more than 1 output


}
/*
void test_1()
{
	for (int i = 0; i < numPatterns / 10; i++)
	{
		patNum = i;
		feedforward();
		cout << "pattern = " << patNum + 1 <<
			"| actual answer = " << trainOutput[patNum] <<
			"| Neural Net guess = " << Guess << endl;
	}
}*/

double getRand(double num1, double num2)
{
	return double((num1 - num2)*(rand() / double(RAND_MAX)) + num1); //(b-a)*random.random() + a
}

double sigmoid(double x)
{
	return 1 / (1 + exp(-x));
}

__device__ double dev_sigmoid(double x)
{
	return 1 / (1 + exp(-x));
}

__device__ double dev_dsigmoid(double x)
{
	return dev_sigmoid(x) * (1 - dev_sigmoid(x));
}

/*void test_2()
{
	int counter = 0;
	for (int i = 0; i < numPatterns / 10; i++)
	{
		patNum = i;
		feedforward();


		if (Guess >= .5 && trainOutput[patNum] == 1)
		{
			counter += 1;
		}
		else if (Guess < .5 && trainOutput[patNum] == -1)
		{
			counter += 1;
		}
	}
	cout << ((counter / (numPatterns / 10)) * 100) << "% classification accuracy" << endl;
}*/

//==================================Kernel Definitions=============================
__global__ void CudaBackProp_part1(double dev_outputDeltas[], double dev_outputActivation[])
{
    int i = threadIdx.x;
	dev_outputDeltas[i] = LR * dev_errThisPat * dev_outputActivation[i];
}

__global__ void CudaBackProp_part2(double dev_outputDeltas[], double**dev_weightsHO,double dev_hidden2Deltas[],double dev_hidden2Activation[])
{
	int j = threadIdx.x;
	double errors = 0;
	for (int k = 0; k < numOutput; k++)
	{
		errors = errors + dev_outputDeltas[k] * dev_weightsHO[j][k];
	}
	dev_hidden2Deltas[j] = dev_dsigmoid(dev_hidden2Activation[j]) * errors;
}

__global__ void CudaBackProp_part3(double dev_hidden2Deltas[], double**dev_weightsH1H2,double dev_hiddenDeltas[],double hiddenActivation[])
{
	int j = threadIdx.x;
	double errors = 0;
	for (int k = 0; k < dev_numHidden2; k++)
	{
		errors = errors + dev_hidden2Deltas[k] * dev_weightsH1H2[j][k];
	}
	dev_hiddenDeltas[j] = dev_dsigmoid(hiddenActivation[j]) * errors;
}

__global__ void CudaBackProp_part4(double dev_outputDeltas[], double dev_hidden2Activation[], double**dev_weightsHO, double**dev_changeOutput)
{
	int j = threadIdx.x;
	double change = 0;
	for (int k = 0; k< numOutput; k++)
	{
		change = dev_outputDeltas[k] * dev_hidden2Activation[j];
		dev_weightsHO[j][k] = dev_weightsHO[j][k] + LR*change + M*dev_changeOutput[j][k];
		dev_changeOutput[j][k] = change;
	}
}

__global__ void CudaBackProp_part5(double dev_hidden2Deltas[], double dev_hiddenActivation[], double**dev_weightsH1H2,double**dev_changeHidden2)
{
	int j = threadIdx.x;
	double change = 0;
	for (int k = 0; k< numHidden2; k++)
	{
		change = dev_hidden2Deltas[k] * dev_hiddenActivation[j];
		dev_weightsH1H2[j][k] = dev_weightsH1H2[j][k] + LR*change + M*dev_changeHidden2[j][k];
		dev_changeHidden2[j][k] = change;
	}
}

__global__ void CudaBackProp_part6(int *patnum, double dev_hiddenDeltas[], double** dev_trainInputs, double**dev_changeHidden, double**dev_weightsIH)
{
	int j = threadIdx.x;
	double change = 0;
	for (int k = 0; k< numHidden; k++)
	{
		change = dev_hiddenDeltas[k] * dev_trainInputs[*patnum][j];
		dev_changeHidden[j][k] = change;
		dev_weightsIH[j][k] = dev_weightsIH[j][k] + LR*change + M*dev_changeHidden[j][k];
	}
}

__global__ void CudaFeedForward_part1(int *patnum, double**dev_trainInputs, double**dev_weightsIH, double dev_hiddenActivation[])
{
	int i = threadIdx.x;
	double SumOf = 0.0;
	for (int j = 0; j < dev_numInputs; j++)
	{
		SumOf = SumOf + (dev_trainInputs[*patnum][j] * dev_weightsIH[j][i]);
	}
	dev_hiddenActivation[i] = dev_sigmoid(SumOf);
}

__global__ void CudaFeedForward_part2(double dev_hiddenActivation[], double**dev_weightsH1H2, double dev_hidden2Activation[])
{
	int i = threadIdx.x;
	double SumOf = 0.0;
	for (int j = 0; j < dev_numHidden; j++)
	{
		SumOf = SumOf + (dev_hiddenActivation[j] * dev_weightsH1H2[j][i]);
	}
	dev_hidden2Activation[i] = dev_sigmoid(SumOf);
}

__global__ void CudaFeedForward_part3(int *patnum, double dev_hidden2Activation[], double**dev_weightsHO, double dev_outputActivation[],double dev_trainOutput[])
{
	int i = threadIdx.x;
	double SumOf = 0.0;
	for (int j = 0; j < dev_numHidden2; j++)
	{
		SumOf = SumOf + (dev_hidden2Activation[j] * dev_weightsHO[j][i]);
	}
	dev_outputActivation[i] = dev_sigmoid(SumOf);
	dev_errThisPat = dev_trainOutput[*patnum] - dev_outputActivation[0];
	dev_RMSerror += dev_errThisPat * dev_errThisPat;
}


